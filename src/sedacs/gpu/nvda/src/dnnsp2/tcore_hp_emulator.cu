#include <iostream>
#include <iomanip>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hipblas.h>

// device function for splitting a float into two halves
__device__ void split_single(const float x, half &hi, half &lo)
{
    hi = __float2half(x);
    float y = (x - __half2float(hi));
    lo = __float2half(y * 1024);
}

// global function for splitting a float matrix into two float halves
template <typename T>
__global__ void array_split_single(const float *AF, half *AH1, half *AH2, const unsigned N)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N)
    {
        half hi;
        half lo;

        split_single(AF[i], hi, lo);

        AH1[i] = hi;
        AH2[i] = lo;
    }
}

void tcoreSPGemmSymm(hipblasHandle_t handle,
                     const unsigned N,
                     const float *A,
                     half *Ah,
                     half *Al,
                     float *B1,
                     float *B2,
                     float *B,
		     hipStream_t *stream)
{
    // Setup kernel launch
    unsigned num_thds = 512;
    unsigned num_blks = int(ceil(float(N * N) / float(num_thds)));

    // Split the floats into the high and low parts
    array_split_single<half><<<num_blks, num_thds>>>(A, Ah, Al, N * N);

    float alpha = 1.0;
    float beta = 0.0;

    // Compute gemmEx for high, set to stream[0]
    hipblasSetStream(handle,stream[0]);
    hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N,
                 &alpha,
                 Ah, HIP_R_16F, N,
                 Ah, HIP_R_16F, N,
                 &beta, B1, HIP_R_32F, N,
                 HIPBLAS_COMPUTE_32F_FAST_16F, HIPBLAS_GEMM_DEFAULT);

    // Compute gemmEx for low, set to stream[1]
    hipblasSetStream(handle,stream[1]);
    hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N,
                 &alpha,
                 Ah, HIP_R_16F, N,
                 Al, HIP_R_16F, N,
                 &beta, B2, HIP_R_32F, N,
                 HIPBLAS_COMPUTE_32F_FAST_16F, HIPBLAS_GEMM_DEFAULT);

    hipStreamSynchronize(stream[0]); hipStreamSynchronize(stream[1]);

    alpha = 1.0;
    beta = 1.0;

    // reset to stream[0]
    hipblasSetStream(handle,stream[0]);
    hipblasSgeam(handle,
                HIPBLAS_OP_N, HIPBLAS_OP_T,
                N, N,
                &alpha,
                B2, N,
                &beta,
                B2, N,
                B, N);

    // undo prior scaling of 2^10
    beta = powf(2, -10);
    hipblasSgeam(handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N,
                N, N,
                &alpha,
                B1, N,
                &beta,
                B, N,
                B, N);
};

void tcoreSPGemmSymm1(hipblasHandle_t handle
                     ,const unsigned N
                     ,const float* A
                     ,const float* B
                     ,half*  Ah
                     ,half*  Al
                     ,half*  Bh
                     ,half*  Bl
                     ,float* C1
                     ,float* C2
                     ,float* C)
{
    // Setup kernel launch
    unsigned MAX_THREADS = 1024;
    unsigned BLOCKS = ceil(N*N/float(MAX_THREADS));
    unsigned THREADS = MAX_THREADS;

    // Split the floats into the high and low parts
    array_split_single<half><<<BLOCKS, THREADS>>>(A, Ah, Al, N*N);

    // Split the floats into the high and low parts
    array_split_single<half><<<BLOCKS, THREADS>>>(B, Bh, Bl, N*N);

    float alpha (1.0f);
    float beta  (0.0f);
    float gamma = powf(2,-10);

    // Set the math mode to allow cuBLAS to use Tensor Cores:
    hipblasStatus_t cublasStat = hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);
    

    // Compute gemm for high
    cublasStat = hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha,
                              Ah, HIP_R_16F, N,
                              Bh, HIP_R_16F, N,
                              &beta, C1, HIP_R_32F, N, HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);

    // Compute gemms for low
    cublasStat = hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha,
                              Ah, HIP_R_16F, N,
                              Bl, HIP_R_16F, N,
                              &beta, C2, HIP_R_32F, N, HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);

    cublasStat = hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha,
                              Al, HIP_R_16F, N,
                              Bh, HIP_R_16F, N,
                              &alpha, C2, HIP_R_32F, N, HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);

    // add the high gemm and low gemm together
    cublasStat = hipblasSgeam(handle,
                             HIPBLAS_OP_N, HIPBLAS_OP_N,
                             N, N,
                             &alpha,
                             C1, N,
                             &gamma,
                             C2, N,
                             C2, N);

    // compute C + C^T 
    cublasStat = hipblasSgeam(handle,
                             HIPBLAS_OP_N, HIPBLAS_OP_T,
                             N, N,
                             &alpha,
                             C2, N,
                             &alpha,
                             C2, N,
                             C, N);

};
