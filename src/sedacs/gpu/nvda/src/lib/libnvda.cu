#include <iostream>
#include <vector>
#include <stdio.h>
#include <lib.h>
#include <structs.h>
#include <diag.cuh>
#include <dnnsp2.cuh>
#include <dnnprt.cuh>
#include <goldensp2.cuh>
#include <movingmusp2.cuh>
#include <pscheby.cuh>
#include <mlsp2.cuh>
#include <error_check.cuh>
#include <hip/hip_fp16.h>

void dm_dnnsp2(double *ham,
               double *dm,
               double *t02,
               float  *id,
               float  *s0,
               float  *s02,
               float  *sbuf1,
               float  *sbuf2,
               void   *vbuf1,
               void   *vbuf2,
               int norb, int nocc, void *handle, hipStream_t *stream)
{

    precision_t u = fp16_fp32;
    refine_t r = yes;
    dnnsp2(ham, dm, t02, id, s0, s02, sbuf1, sbuf2, vbuf1, vbuf2, norb, nocc, u, r, handle, stream);
}

/*void dm_finitetsp2(double *ham,
                  double *dm,
                  int norb, double mu,
                  void *handle)
{
    precision_t u = fp16_fp32;
    refine_t r = no;
    finitetsp2(ham, dm, norb, mu, u, r, handle);
}
*/
void dm_goldensp2(double *ham,
                  double *dm,
                  int norb, double mu,
                  void *handle)
{
    precision_t u = fp16_fp32;
    refine_t r = no;
    goldensp2(ham, dm, norb, mu, u, r, handle);
}

void dm_movingmusp2(double *ham,
                    double *dm,
                    int norb, double mu,
                    void *handle)
{
    precision_t u = fp16_fp32;
    refine_t r = no;
    movingmusp2(ham, dm, norb, mu, u, r, handle);
}

void dm_dnnprt(double *ham, double *prt,
               double *dm, double *rsp,
               int norb, int nocc, void *handle)
{

    // precision_t u = fp16_fp32;
    // refine_t r = yes;

    dnnprt(ham, prt, dm, rsp, norb, nocc); //, handle);
}

void dm_mlsp2(double *model, double *ham, double *dm, int nlayers, int norb)
{
    precision_t u = fp16_fp32;
    refine_t r = yes;

    mlsp2(model, ham, dm, nlayers, norb, u, r);
}

void dm_pscheby(double *ham, double *dm, int norb, int nocc, double kbt)
{

    double bndfil = 0.666666;

    precision_t u = fp64;

    refine_t r = yes;

    int K = 32;
    int M = 32;

    pscheby(ham, dm, K, M, norb, nocc, kbt);
}

void involap(double *overlap, double *guess, double *factor, int norb)
{

     precision_t u = fp64;
     refine_t r = yes;

    // invOlapFactorize(ham, dm, kbt, bndfil, u, norb, nocc);
}

void dm_diag(double *ham, double *dm, double kbt, int norb, int nocc, double bndfil)
{

    precision_t u = fp64;
    refine_t r = yes;

    diagonalize(ham, dm, kbt, bndfil, u, norb, nocc);
}

/*
    Wrap hipMalloc with python
*/
void *dev_alloc(size_t size)
{

    double *devptr;
    CUDA_CHECK_ERR(hipMalloc(&devptr, size));

    std::cout << devptr << std::endl;

    return (void *)devptr;
}

/*
    Wrap hipSetDevice with python
*/
void set_device(int device)
{
    CUDA_CHECK_ERR(hipSetDevice(device));

    std::cout << "Device set to " << device << std::endl;
}

/*
    Wrap hipStreamCreate with python
*/
void *set_stream(void)
{
    hipStream_t *stream = (hipStream_t*) malloc(sizeof(hipStream_t) * 10);

    // create 10 cuda streams
    for (int i=0; i<10;i++){
	    CUDA_CHECK_ERR(hipStreamCreate(&stream[i]));
    }
    
    std::cout << stream << std::endl;

    return (void*) stream;
}

/*
    Initalize cublas handle with python
*/
void *cublasInit()
{
    hipblasHandle_t *handle = (hipblasHandle_t*) malloc(sizeof(hipblasHandle_t));
    CUBLAS_CHECK_ERR(hipblasCreate(handle));
    
    // Set math mode
    CUBLAS_CHECK_ERR(hipblasSetMathMode(*handle, HIPBLAS_DEFAULT_MATH));
    std::cout << "cublas handle intialized" << std::endl;

    return (void *) handle;
}

/*
    Destroy cublas handle with python
*/
/*void hipblasDestroy(hipblasHandle_t *handle)
{
    CUBLAS_CHECK_ERR(hipblasDestroy(*handle));
    
    // Set math mode
    std::cout << "cublas handle destroyed" << std::endl;
}
*/

/*
    Wrap hipGetDevice with python
*/
int get_device()
{
    int device = 0;

    CUDA_CHECK_ERR(hipGetDevice(&device));

    return device;
}


/*
    Wrap hipMallocManaged with python
*/
void *dev_alloc_managed(size_t size)
{

    double *devptr;
    CUDA_CHECK_ERR(hipMallocManaged(&devptr, size));

    std::cout << devptr << std::endl;

    return (void *)devptr;
}

/*
    Wrap hipHostMalloc with python
*/
void *host_alloc_pinned(size_t size)
{

    double *hostptr;
    CUDA_CHECK_ERR(hipHostMalloc(&hostptr, size));

    std::cout << hostptr << std::endl;

    return (void *)hostptr;
}

/*
    Wrap hipMemcpy with python, Host to Host
*/
void memcpyHtoH(void *dest, void *source, size_t size)
{

    CUDA_CHECK_ERR(hipMemcpy(dest, source, size, hipMemcpyHostToHost));

    std::cout << "Memcpy H to H" << std::endl;
}

/*
    Wrap hipMemcpy with python, Device to Host
*/
void memcpyDtoH(void *dest, void *source, size_t size)
{

    CUDA_CHECK_ERR(hipMemcpy(dest, source, size, hipMemcpyDeviceToHost));

    std::cout << "Memcpy D to H" << std::endl;
}

/*
    Wrap hipMemcpy with python, Host to Device
*/
void memcpyHtoD(void *dest, void *source, size_t size)
{

    CUDA_CHECK_ERR(hipMemcpy(dest, source, size, hipMemcpyHostToDevice));

    std::cout << "Memcpy H to D" << std::endl;
}

/*
    Wrap hipMemcpyAsync with python, Device to Host
*/
void memcpyasyncDtoH(void *dest, void *source, size_t size)
{

    CUDA_CHECK_ERR(hipMemcpyAsync(dest, source, size, hipMemcpyDeviceToHost));

    std::cout << "Memcpy Async" << std::endl;
}


/*
    Wrap hipMemcpyAsync with python, Host to Device
*/
void memcpyasyncHtoD(void *dest, void *source, size_t size)
{

    CUDA_CHECK_ERR(hipMemcpyAsync(dest, source, size, hipMemcpyHostToDevice));

    std::cout << "Memcpy Async" << std::endl;
}


/*
    Wrap hipFree with python
*/
void dev_free(void *devptr)
{

    std::cout << devptr << std::endl;
    CUDA_CHECK_ERR(hipFree(devptr));
    std::cout << "Free" << std::endl;
}

